#include "stdio.h"
#include "hip/hip_runtime.h"
#include ""
#include "functions.h"
#include "trie.h"
#include "generator.h"
#include "bruteforce.h"

int main()
{ 
	
	hipError_t cudaStatus;
	//time structures
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsedTime;
	
	///////////////////////////// INIT IPS AND MASKS ///////////////////////////
	//init ips and masks
	u_char *ips= (u_char*)malloc(NUM_IPS*IPV4_B*sizeof(u_char));
	u_char *masks = (u_char*)malloc(NUM_MASKS*IPV4M_B* sizeof(u_char));

	printf("mem for IPs and MASKs allocated.\n");
	////////////////////////////////////////////////////////////////////////////

	//warmup by empty kernel
	init();

	generate_ip_addresses(ips);
	printf("IPs generated on CPU");
	generate_masks(masks);
	printf("Masks generated on CPU");
	////////////////////////////// BRUTE FORCE //////////////////////////////////
	u_char *assignedMasks = (u_char*)malloc(NUM_IPS * IPV4M_B * sizeof(u_char));
	hipEventRecord(start);
	bruteforce(ips, masks, assignedMasks, NUM_IPS*IPV4_B, NUM_MASKS*IPV4M_B);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("time elapsed: %f\n", elapsedTime);
	wrtiteToFile(ips, assignedMasks);
	////////////////////////////////////////////////////////////////////////////

	//Array for determining the first byte of mask (limit tree nodes)
	/*u_char *byteMaskArr = (u_char*)malloc(U_CHAR_SIZE*sizeof(u_char));
	getByteMaskArr(byteMaskArr, masks, 1); //sets up array of used ipmasks (their first Byte)
	u_char no_children_node = countUniqueMaskBytes(byteMaskArr);

	TrieNode *root = (TrieNode *)malloc(sizeof(TrieNode));
	root = create_trienode(0, 0, 0,no_children_node);
	createTrie(root, masks);
	//here assign ips to masks - tree traversing
	destroy_trienode(root);
	
	//count and take values of nodes from this
	printByteMaskArr(byteMaskArr);*/

	free(ips);
	free(masks);
	//free(byteMaskArr);
	free(assignedMasks);
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

